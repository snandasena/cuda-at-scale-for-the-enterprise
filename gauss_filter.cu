//
// Created by sajith on 17/11/2024.
//

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif


#include <UtilNPP/Exceptions.h>
#include <UtilNPP/ImagesCPU.h>
#include <UtilNPP/ImagesNPP.h>
#include <UtilNPP/ImageIO.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nppi.h>

#include <string>

bool printfNPPinfo(int argc, char *argv[])
{
    const NppLibraryVersion *libVer = nppGetLibVersion();

    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor,
           libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  CUDA Driver  Version: %d.%d\n", driverVersion / 1000,
           (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000,
           (runtimeVersion % 100) / 10);

    // Min spec is SM 1.0 devices
    bool bVal = checkCudaCapabilities(1, 0);
    return bVal;
}


void gaussFilter(const std::string &filePath, const std::string &outputFile)
{
    try
    {
        std::cout << "Processing of " << filePath << " started." << std::endl;
        npp::ImageCPU_8u_C1 hostSrc;
        npp::loadImage(filePath, hostSrc);
        npp::ImageNPP_8u_C1 deviceSrc(hostSrc);
        const NppiSize srcSize = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        const NppiPoint srcOffset = {0, 0};

        const NppiSize filterROI = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        npp::ImageNPP_8u_C1 deviceDst(filterROI.width, filterROI.height);

        NPP_CHECK_NPP(nppiFilterGaussBorder_8u_C1R(deviceSrc.data(), deviceSrc.pitch(), srcSize, srcOffset,
                                                   deviceDst.data(), deviceDst.pitch(), filterROI,
                                                   NppiMaskSize::NPP_MASK_SIZE_3_X_3,
                                                   NppiBorderType::NPP_BORDER_REPLICATE));

        npp::ImageCPU_8u_C1 hostDst(deviceDst.size());
        deviceDst.copyTo(hostDst.data(), hostDst.pitch());
        npp::saveImage(outputFile, hostDst);
        std::cout << "Processing of " << filePath << " ended. Result saved to: " << outputFile << std::endl;

//        nppiFree(deviceSrc.data());
//        nppiFree(deviceDst.data());
//        nppiFree(hostSrc.data());
//        nppiFree(hostDst.data());
    }
    catch (npp::Exception &rException)
    {
        std::cerr << "Program error! The following exception occurred: \n";
        std::cerr << rException << std::endl;
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
    }
    catch (...)
    {
        std::cerr << "Program error! An unknow type of exception occurred. \n";
        std::cerr << "Aborting." << std::endl;

        exit(EXIT_FAILURE);
    }
}


int main(int argc, char *argv[])
{

    if (!printfNPPinfo(argc, argv))
    {
        exit(EXIT_SUCCESS);
    }

    findCudaDevice(argc, (const char **) argv);

    return 0;
}