#include <UtilNPP/Exceptions.h>
#include <UtilNPP/ImagesCPU.h>
#include <UtilNPP/ImagesNPP.h>
#include <UtilNPP/ImageIO.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <nppi.h>

#include <string>
#include <iostream>
#include <vector>
#include <thread>
#include <filesystem>

namespace fs = std::filesystem;

/**
 * Prints detailed information about the NPP library and CUDA driver versions.
 *
 * This function retrieves and displays the version information for the NPP (NVIDIA Performance Primitives) library,
 * the CUDA driver, and the CUDA runtime. It also checks if the system meets the minimum CUDA capabilities.
 *
 * @param argc Number of command-line arguments passed to the program.
 * @param argv Array of command-line arguments.
 * @return True if the system meets the minimum CUDA capability requirements, otherwise false.
 */
bool printfNPPinfo(int argc, char *argv[])
{
    const NppLibraryVersion *libVer = nppGetLibVersion();
    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    printf("  CUDA Driver Version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    return checkCudaCapabilities(1, 0);  // Checks for a minimum of Compute Capability 1.0
}

/**
 * Applies a Gaussian filter to an input image and saves the processed result to an output file.
 *
 * This function uses the NVIDIA Performance Primitives (NPP) library to apply a Gaussian filter
 * to the input image. The processed image is then saved to the specified output file location.
 *
 * @param filePath Path to the input image file.
 * @param outputFile Path to save the filtered image.
 * @throws npp::Exception if any errors occur during NPP operations.
 * @throws std::exception for any other runtime errors.
 */
void applyGaussFilter(const std::string &filePath, const std::string &outputFile)
{
    try
    {
        std::cout << "Processing: " << filePath << std::endl;

        // Load the input image into CPU memory
        npp::ImageCPU_8u_C1 hostSrc;
        npp::loadImage(filePath, hostSrc);

        // Copy the image to GPU memory
        npp::ImageNPP_8u_C1 deviceSrc(hostSrc);

        const NppiSize srcSize = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        const NppiPoint srcOffset = {0, 0};
        const NppiSize filterROI = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        npp::ImageNPP_8u_C1 deviceDst(filterROI.width, filterROI.height);

        // Apply Gaussian filter using NPP
        NPP_CHECK_NPP(nppiFilterGaussBorder_8u_C1R(deviceSrc.data(), deviceSrc.pitch(), srcSize, srcOffset,
                                                   deviceDst.data(), deviceDst.pitch(), filterROI,
                                                   NppiMaskSize::NPP_MASK_SIZE_3_X_3,
                                                   NppiBorderType::NPP_BORDER_REPLICATE));

        // Copy the processed image back to CPU memory
        npp::ImageCPU_8u_C1 hostDst(deviceDst.size());
        deviceDst.copyTo(hostDst.data(), hostDst.pitch());

        // Save the processed image to the output file
        npp::saveImage(outputFile, hostDst);
        std::cout << "Finished: " << filePath << " -> " << outputFile << std::endl;

        // Free allocated memory
        nppiFree(deviceSrc.data());
        nppiFree(deviceDst.data());
        nppiFree(hostSrc.data());
        nppiFree(hostDst.data());
    }
    catch (const std::exception &e)
    {
        std::cerr << "Error processing " << filePath << ": " << e.what() << std::endl;
    }
    catch (...)
    {
        std::cerr << "Unknown error processing " << filePath << std::endl;
    }
}

/**
 * Cleans up the output directory by deleting all files in it.
 *
 * @param outputDir Path to the directory to clean up.
 * @throws std::filesystem::filesystem_error if any file operations fail.
 */
void cleanupOutputDirectory(const std::string &outputDir)
{
    for (const auto &entry: fs::directory_iterator(outputDir))
    {
        if (entry.is_regular_file())
        {
            std::cout << "Deleting: " << entry.path() << std::endl;
            fs::remove(entry.path());
        }
    }
}

/**
 * Processes a batch of images concurrently using threads.
 *
 * This function takes a batch of image paths and processes them using threads, with each thread
 * applying the Gaussian filter to one image. All threads are joined before the function exits.
 *
 * @param batch A vector of paths to the image files in the batch.
 * @param outputDir Directory to save the processed images.
 */
void processBatch(const std::vector<std::string> &batch, const std::string &outputDir)
{
    std::vector<std::thread> workers;
    for (const auto &imagePath: batch)
    {
        if (fs::exists(imagePath) && fs::is_regular_file(imagePath))
        {
            workers.emplace_back([=]()
                                 {
                                     std::string outputFile =
                                             outputDir + "/" + imagePath.substr(imagePath.find_last_of("/\\") + 1);
                                     outputFile =
                                             outputFile.substr(0, outputFile.find_last_of(".")) + "_gauss_filtered.pgm";
                                     applyGaussFilter(imagePath, outputFile);
                                 });
        }
        else
        {
            std::cerr << "Invalid image path: " << imagePath << std::endl;
        }
    }

    // Wait for all threads in the batch to complete
    for (auto &worker: workers)
    {
        worker.join();
    }
}

/**
 * Processes all images in a directory in batches.
 *
 * This function iterates over all images in the input directory, organizes them into batches,
 * and processes each batch concurrently using `processBatch`. The number of threads used
 * is determined by the number of CPU cores.
 *
 * @param inputDir Directory containing the input images.
 * @param outputDir Directory to save the processed images.
 */
void processImagesInDirectory(const std::string &inputDir, const std::string &outputDir)
{
    if (fs::exists(outputDir))
        cleanupOutputDirectory(outputDir);
    else
        fs::create_directory(outputDir);

    // Collect all valid image paths from the input directory
    std::vector<std::string> imagePaths;
    for (const auto &entry: fs::recursive_directory_iterator(inputDir))
    {
        imagePaths.push_back(entry.path().string());
    }

    // Determine the number of threads to use
    unsigned int numThreads = std::thread::hardware_concurrency();
    std::vector<std::string> batch;

    // Process images in batches
    for (size_t i = 0; i < imagePaths.size(); ++i)
    {
        batch.push_back(imagePaths[i]);

        if (batch.size() == numThreads || i == imagePaths.size() - 1)
        {
            processBatch(batch, outputDir);
            batch.clear();
        }
    }
}


// Function to parse input and output directories from command-line arguments
std::tuple<std::string, std::string> parseInputOutputDirs(int argc, char *argv[])
{
    // Default input and output directories
    std::string inputDir = "../data/";
    std::string outputDir = "../output/";

    // Parse command-line arguments for input and output directories
    if (checkCmdLineFlag(argc, (const char **) argv, "input"))
    {
        char *inputPath = nullptr;
        getCmdLineArgumentString(argc, (const char **) argv, "input", &inputPath);
        if (inputPath)
        {
            inputDir = inputPath;
        }
    }

    if (checkCmdLineFlag(argc, (const char **) argv, "output"))
    {
        char *outputPath = nullptr;
        getCmdLineArgumentString(argc, (const char **) argv, "output", &outputPath);
        if (outputPath)
        {
            outputDir = outputPath;
        }
    }

    // Return the directories as a tuple
    return std::make_tuple(inputDir, outputDir);
}


/**
 * Entry point for the program.
 *
 * This function prints CUDA and NPP version information, then processes all images
 * in the specified input directory by applying a Gaussian filter. The processed images
 * are saved to the specified output directory.
 */
int main(int argc, char *argv[])
{
    if (!printfNPPinfo(argc, argv))
    {
        exit(EXIT_SUCCESS);
    }


    // Call the function to parse input and output directories
    std::string inputDir, outputDir;
    std::tie(inputDir, outputDir) = parseInputOutputDirs(argc, argv);

    // Log the directories being used
    std::cout << "Input Directory: " << inputDir << std::endl;
    std::cout << "Output Directory: " << outputDir << std::endl;

    // Process images in the input directory
    processImagesInDirectory(inputDir, outputDir);

    return 0;
}
