//
// Created by sajith on 17/11/2024.
//
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#pragma warning(disable : 4819)
#endif

#include <UtilNPP/Exceptions.h>
#include <UtilNPP/ImagesCPU.h>
#include <UtilNPP/ImagesNPP.h>
#include <UtilNPP/ImageIO.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <nppi.h>

#include <string>
#include <iostream>
#include <vector>
#include <thread>
#include <chrono>
#include <filesystem>
#include <mutex>
#include <queue>
#include <atomic>
#include <condition_variable>

namespace fs = std::filesystem;  // For filesystem operations like iterating over files

/**
 * Prints NPP library and CUDA driver information.
 * This function retrieves the version of the NPP library, CUDA driver, and runtime,
 * and checks if the system meets the minimum CUDA capability (SM 1.0).
 *
 * @param argc Argument count.
 * @param argv Argument values (command-line arguments).
 * @return True if the system supports the minimum CUDA capability, otherwise false.
 */
bool printfNPPinfo(int argc, char *argv[])
{
    const NppLibraryVersion *libVer = nppGetLibVersion();

    // Print NPP library version
    printf("NPP Library Version %d.%d.%d\n", libVer->major, libVer->minor, libVer->build);

    int driverVersion, runtimeVersion;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);

    // Print CUDA driver and runtime versions
    printf("  CUDA Driver Version: %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
    printf("  CUDA Runtime Version: %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);

    // Check CUDA capabilities for SM 1.0 devices
    bool bVal = checkCudaCapabilities(1, 0);
    return bVal;
}

/**
 * Applies a Gaussian filter to an image and saves the result to an output file.
 * The function loads the image from the specified file path, applies the Gaussian filter
 * using NPP, and saves the filtered result to the output file.
 *
 * @param filePath Path to the input image file.
 * @param outputFile Path to save the processed image.
 */
void applyGaussFilter(const std::string &filePath, const std::string &outputFile)
{
    try
    {
        std::cout << "Processing of " << filePath << " started." << std::endl;

        // Load image into CPU memory
        npp::ImageCPU_8u_C1 hostSrc;
        npp::loadImage(filePath, hostSrc);

        // Copy image to GPU memory
        npp::ImageNPP_8u_C1 deviceSrc(hostSrc);

        const NppiSize srcSize = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        const NppiPoint srcOffset = {0, 0};

        const NppiSize filterROI = {(int) deviceSrc.width(), (int) deviceSrc.height()};
        npp::ImageNPP_8u_C1 deviceDst(filterROI.width, filterROI.height);

        // Apply Gaussian filter using NPP
        NPP_CHECK_NPP(nppiFilterGaussBorder_8u_C1R(deviceSrc.data(), deviceSrc.pitch(), srcSize, srcOffset,
                                                   deviceDst.data(), deviceDst.pitch(), filterROI,
                                                   NppiMaskSize::NPP_MASK_SIZE_3_X_3,
                                                   NppiBorderType::NPP_BORDER_REPLICATE));

        // Copy filtered image back to CPU memory
        npp::ImageCPU_8u_C1 hostDst(deviceDst.size());
        deviceDst.copyTo(hostDst.data(), hostDst.pitch());

        // Save the processed image to output file
        npp::saveImage(outputFile, hostDst);

        std::cout << "Processing of " << filePath << " ended. Result saved to: " << outputFile << std::endl;

        // Free memory allocated for both input and output images
        nppiFree(deviceSrc.data());
        nppiFree(deviceDst.data());
        nppiFree(hostSrc.data());
        nppiFree(hostDst.data());
    }
    catch (npp::Exception &rException)
    {
        std::cerr << "Program error! The following exception occurred: \n";
        std::cerr << rException << std::endl;
        std::cerr << "Aborting." << std::endl;
        exit(EXIT_FAILURE);
    }
    catch (...)
    {
        std::cerr << "Program error! An unknown type of exception occurred. \n";
        std::cerr << "Aborting." << std::endl;
        exit(EXIT_FAILURE);
    }
}

/**
 * Thread worker that processes images by applying the Gaussian filter.
 *
 * @param taskQueue The queue of tasks (image paths) to process.
 * @param done A flag to signal when processing is complete.
 * @param outputDir The directory where processed images will be saved.
 */
void processImageWorker(std::queue<std::string> &taskQueue, std::mutex &queueMutex, std::condition_variable &cv,
                        std::atomic<bool> &done, const std::string &outputDir)
{
    while (true)
    {
        std::string imagePath;
        {
            std::unique_lock<std::mutex> lock(queueMutex);

            // Wait until there is work to do
            cv.wait(lock, [&] { return !taskQueue.empty() || done; });

            if (taskQueue.empty() && done)
                break;

            // Get the next image path from the queue
            imagePath = taskQueue.front();
            taskQueue.pop();
        }

        // Generate output file path
        std::string outputFile = outputDir + "/" + imagePath.substr(imagePath.find_last_of("/\\") + 1);
        outputFile = outputFile.substr(0, outputFile.find_last_of(".")) + "-filtered.pgm";

        // Process the image
        applyGaussFilter(imagePath, outputFile);
    }
}

/**
 * Cleans up the output directory by deleting all files in it.
 *
 * @param outputDir Path to the output directory.
 */
void cleanupOutputDirectory(const std::string &outputDir)
{
    for (const auto &entry : fs::directory_iterator(outputDir))
    {
        if (entry.is_regular_file())
        {
            std::cout << "Deleting file: " << entry.path() << std::endl;
            fs::remove(entry.path());
        }
    }
}

/**
 * Processes all images in a directory with a maximum number of threads equal to the CPU cores.
 *
 * @param inputDir Path to the input directory containing images.
 * @param outputDir Path to save the processed images.
 */
void processImagesInDirectory(const std::string &inputDir, const std::string &outputDir)
{
    // Clean up the output directory before starting processing
    cleanupOutputDirectory(outputDir);

    std::vector<std::thread> workers;
    std::queue<std::string> taskQueue;
    std::mutex queueMutex;
    std::condition_variable cv;
    std::atomic<bool> done(false);

    // Discover all image files in the input directory
    for (const auto &entry : fs::directory_iterator(inputDir))
    {
        if (entry.is_regular_file() && (entry.path().extension() == ".bmp" || entry.path().extension() == ".jpg"))
        {
            taskQueue.push(entry.path().string());
        }
    }

    // Start worker threads (limit to the number of available CPU cores)
    unsigned int numThreads = std::thread::hardware_concurrency();
    for (unsigned int i = 0; i < numThreads; ++i)
    {
        workers.push_back(std::thread(processImageWorker, std::ref(taskQueue), std::ref(queueMutex), std::ref(cv),
                                      std::ref(done), std::ref(outputDir)));
    }

    // Notify workers to start processing
    {
        std::unique_lock<std::mutex> lock(queueMutex);
        done = false; // Set done to false as we are processing tasks
    }
    cv.notify_all();

    // Wait for all workers to finish processing
    for (auto &worker : workers)
    {
        worker.join();
    }

    // Mark that processing is done and notify all workers to exit
    {
        std::unique_lock<std::mutex> lock(queueMutex);
        done = true;
    }
    cv.notify_all();
}

int main(int argc, char *argv[])
{
    // Print NPP and CUDA version information
    if (!printfNPPinfo(argc, argv))
    {
        exit(EXIT_SUCCESS);
    }

    // Input directory containing images
    std::string inputDir = "/home/sajith/dev/cuda-at-scale-for-the-enterprise/data/";

    // Output directory where processed images will be saved
    std::string outputDir = "/home/sajith/dev/cuda-at-scale-for-the-enterprise/output/";

    // Process images in the input directory
    processImagesInDirectory(inputDir, outputDir);

    return 0;
}
